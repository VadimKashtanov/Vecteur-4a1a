#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_template/tmpl_etc.cu"
/*
__global__
static void kerd_lire(float * p__d, uint p, float * val) {
	val[0] = p__d[p];
};

static float lire(float * p__d, uint p) {
	float * val = cudalloc<float>(1);
	kerd_lire<<<1,1>>>(p__d, p, val);
	ATTENDRE_CUDA();
	//
	float * _ret = gpu_vers_cpu<float>(val, 1);
	float ret = _ret[0];
	free(_ret);cudafree<float>(val);
	//
	return ret;
};

static float ** toutes_les_predictions(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	//
	uint I = btcusdt->I;
	uint T = btcusdt->T;
	uint L = btcusdt->L;
	uint N = btcusdt->N;
	//
	float ancien_u = 100.0;
	float u = 100.0;
	//
	ASSERT(btcusdt->T % MEGA_T == 0);
	//
	uint _T     = (btcusdt->T - (btcusdt->T % MEGA_T))/MEGA_T;
	uint PREDS = _T * MEGA_T;
	//
	float * les_predictions = alloc<float>(PREDS);
	float * les_deltas      = alloc<float>(PREDS);
	float * les_prixs       = alloc<float>(PREDS);
	
	//
	uint lp = 0;
	//
	printf("[t=0] u = %f $\n", u);
	FOR(0, _t_, _T) {
		//
		uint ts[GRAND_T];
		FOR(0, t, GRAND_T) ts[t] = _t_*MEGA_T + 0;
		//
		uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
		
		//
		mdl_f(mdl, btcusdt, ts__d, false);
		//
		uint Y    = mdl->inst[mdl->sortie]->Y;
		float * y = gpu_vers_cpu<float>(mdl->inst[mdl->sortie]->y__d, GRAND_T*MEGA_T*Y);
		//
		FOR(0, mega_t, MEGA_T) {
			uint ty = t_MODE(0, mega_t);
			//
			uint pos = _t_*MEGA_T + mega_t;
			float p0 = lire(btcusdt->prixs__d, _t_  );
			float p1 = (_t_ == _T-1 ? p0 : lire(btcusdt->prixs__d, _t_+1));
			//
			les_predictions[pos] = y[ty*Y + 0];
			les_deltas     [pos] = p1/p0 - 1.0;
			les_prixs      [pos] = p0;
			//
			u += u * 10 * les_predictions[pos] * les_deltas[pos];
			if (u < 0) u = 0;
		}

		//
		cudafree<uint>(ts__d);
		free(y);
		printf("[t=%i] u = %f $ ", 1+_t_, u);
		if      (ancien_u > u) printf("\033[91m-%.2g$\033[0m", abs(ancien_u-u));
		else if (ancien_u < u) printf("\033[92m+%.2g$\033[0m", abs(ancien_u-u));
		else                   printf("\033[2m  ?\033[0m");
		printf("\n");
		ancien_u = u;
	};
	//
	float ** ret = alloc<float*>(3);
	ret[0] = les_predictions;
	ret[1] = les_deltas     ;
	ret[2] = les_prixs      ;
	return ret;
};*/

int main() {
/*	srand(0);
	init_listes_instructions();
	ecrire_structure_generale("structure_generale.bin");
	verif_insts();

	//	=========================================================
	//	=========================================================
	//	=========================================================
	BTCUSDT_t * btcusdt = cree_btcusdt("prixs/tester_model_donnee.bin");

	//	=========================================================
	//	=========================================================
	//	=========================================================

	//	--- Mdl_t ---
	Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	float ** __lp = toutes_les_predictions(mdl, btcusdt);
	float * preds  = __lp[0];
	float * deltas = __lp[1];
	float * prixs  = __lp[2];

	FILE * fp = FOPEN("les_predictions.bin", "wb");
	//
	uint T     = (btcusdt->T - (btcusdt->T % MEGA_T))/MEGA_T;
	uint PREDS = T * MEGA_T;
	//
	FWRITE(preds, sizeof(float), PREDS, fp);	//les prédictions
	free(preds);
	//
	FWRITE(deltas, sizeof(float), PREDS, fp);	//les déltas
	free(deltas);
	//
	FWRITE(prixs, sizeof(float), PREDS, fp);	//les déltas
	free(prixs);
	//
	fclose(fp);

	//	=========================================================
	//	=========================================================
	//	=========================================================
	//
	//plumer_le_score(mdl, btcusdt);

	//
	liberer_mdl    (mdl    );
	liberer_btcusdt(btcusdt);*/
};