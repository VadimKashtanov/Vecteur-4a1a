#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__f_df_btcusdt(
	float * S,
	//
	float * prixs,
	//
	float * y, float * dy,
	float * w,
	uint * ts__d,
	//
	uint T, uint Y)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x; 
	//uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//uint  i = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (_y < Y) {
		float s = 0;
		FOR(0, _t, GRAND_T) {
			FOR(0, mega_t, MEGA_T) {
				uint ty        = t_MODE(_t, mega_t);
				uint t_btcusdt = ts__d[_t] + 1 + mega_t;
				//
				assert(t_btcusdt != T-1);	//le dernier bloque n'a pas de y car on connait pas le future
				assert(t_btcusdt  < T  );	//verifier qu'il existe
				//
				uint wpos = t_btcusdt*Y + 0; float _w = w[wpos];
				uint ypos = ty       *Y + 0; float _y = y[ypos];
				//
				float K = powf(prixs[t_btcusdt+1]/prixs[t_btcusdt] - 1, P_COEF);
				//
				if (_y != _y) {
					printf("ypos=%i y=%f t_btcusdt=%i ty=%i T=%i dernier t=%i\n", ypos, _y, t_btcusdt, ty, T, T-1);
					assert(0);
				}
				assert(_y >= -100 && _y <= +100);
				//
				float coef = K / (float)(GRAND_T * MEGA_T * Y);
				s       += ( score_p2(_y, _w, 2)) * coef;
				float ds = (dscore_p2(_y, _w, 2)) * coef;
				//
				dy[ty*Y + ypos] = ds;
			}
		}
		//
		atomicAdd(&S[0], s);
	}
};

float f_df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	float * S__d = cudalloc<float>(1);
	//
	k__f_df_btcusdt<<<dim3(KERD(btcusdt->Y, 32)), dim3(32)>>>(
		S__d,
		//
		btcusdt->prixs__d,
		//
		y__d, dy__d,
		btcusdt->y__d,
		ts__d,
		//
		btcusdt->T, btcusdt->Y
	);
	ATTENDRE_CUDA();
	//
	//
	float * S = gpu_vers_cpu<float>(S__d, 1);
	float s = S[0] / (float)(MEGA_T*GRAND_T*btcusdt->Y);
	//
	cudafree<float>(S__d);
	    free       (S   );
	//
	return s;
};