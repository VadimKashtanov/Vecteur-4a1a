#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__f_df_btcusdt(
	float * S,
	//
	float * prixs,
	//
	float * y, float * dy,
	float * w,
	uint * ts__d,
	//
	uint T, uint Y)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x; 
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	uint mega_t = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (_y < Y && _t < GRAND_T) {
		float s = 0;
		FOR(0, mega_t, MEGA_T) {
			uint ty        = t_MODE(_t, mega_t);
			uint t_btcusdt = ts__d[_t] + mega_t;//+ 1 + mega_t;
			//
			//printf("%i %i _t=%i\n", t_btcusdt, ts__d[_t], _t);
			assert(t_btcusdt != T-1);	//le dernier bloque n'a pas de y car on connait pas le future
			assert(t_btcusdt  < T  );	//verifier qu'il existe
			//
			uint wpos = t_btcusdt*Y + _y; float __w = w[wpos];
			uint ypos = ty       *Y + _y; float __y = y[ypos];
			//
			float K = powf(100*fabs(prixs[t_btcusdt+1]/prixs[t_btcusdt] - 1), P_COEF);
			//
			if (__y != __y) {
				printf("ypos=%i _y=%i y=%f t_btcusdt=%i ty=%i T=%i dernier t=%i\n", ypos, (ypos-ty*Y), __y, t_btcusdt, ty, T, T-1);
				assert(0);
			}
			if (!(__y >= -100 && __y <= +100)) printf("__y=%f ypos=%i+%i\n", __y, ty*Y, _y);
			assert(__y >= -100 && __y <= +100);
			//
			float coef = K / (float)(GRAND_T * MEGA_T * Y);
			s       += ( score_p2(__y, __w, 2)) * coef;// * (sng(__y)==sng(__w) ? 1:2);
			float ds = (dscore_p2(__y, __w, 2)) * coef;// * (sng(__y)==sng(__w) ? 1:2);
			//
			dy[ypos] = ds;
		}
		//
		atomicAdd(&S[0], s);
	}
};

float f_df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	float * S__d = cudalloc<float>(1);
	//
	k__f_df_btcusdt<<<dim3(KERD(btcusdt->Y, 16), KERD(GRAND_T, 16), KERD(MEGA_T,1)), dim3(16,16,1)>>>(
		S__d,
		//
		btcusdt->prixs__d,
		//
		y__d, dy__d,
		btcusdt->y__d,
		ts__d,
		//
		btcusdt->T, btcusdt->Y
	);
	ATTENDRE_CUDA();
	//
	//
	float * S = gpu_vers_cpu<float>(S__d, 1);
	float s = S[0];// / (float)(MEGA_T*GRAND_T*btcusdt->Y);
	//
	cudafree<float>(S__d);
	    free       (S   );
	//
	return s;
};