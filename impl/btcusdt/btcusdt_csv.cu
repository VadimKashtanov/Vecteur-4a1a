#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static char * lire_chars(FILE * fp) {
	fseek(fp, 0, SEEK_END);
	uint len = ftell(fp);
	ASSERT(len != 0);
	fseek(fp, 0, SEEK_SET);
	char * str = (char*)malloc(len+1);
	FREAD(str, 1, len, fp);
	str[len-1+1] = '\0';
	return str;
};

static uint nombre_lignes(char * txt) {
	uint _lignes = 0;
	uint pos = 0;
	while ( txt[pos] != '\0') {
		if (txt[pos] == '\n') _lignes++;
		pos++;
	}
	return _lignes+1; // car la dernière se termine avec un '\0'
};

static char *** tableau_csv(char * txt) {
	//	-- Parser le CSV --
	uint lignes = nombre_lignes(txt);
	char ** ligne = split(txt, '\n');//texte_vers_lignes(txt, lignes);
	//FOR(0, i, lignes) printf("%s\n", ligne[i]);

	//	-- Traiter le CSV --
	free(ligne[0]); free(ligne[1]);	//https ...\n Hight,low,close...
	lignes -= 2;
	FOR(0, i, lignes) ligne[i] = ligne[i+2];
	//
	char *** tableau = (char***)malloc(sizeof(char**) * lignes); //lignes_vers_tableau(ligne, lignes);
	FOR(0, i, lignes) {
		tableau[i] = split(ligne[i], ',');
		//FOR(0, m, MOTS) printf("[%s]", tableau[i][m]);
		//printf("\n");
		free(ligne[i]);
	}
	free(ligne);
	//
	return tableau;
};

//	--------------------------------------------------

typedef struct {
	char  *  nom;
	float * ligne;
	uint T;
} Ligne_t;

static Ligne_t * extraire(char * nom, char *** tableau, uint T, uint mot) {
	Ligne_t * ligne = alloc<Ligne_t>(1);
	ligne->nom = nom;
	ligne->ligne = alloc<float>(T);
	ligne->T = T;
	FOR(0, i, T) ligne->ligne[i] = atof(tableau[T-1-i][mot]); // la 0-eme est la plus récente
	return ligne;
};

Ligne_t ** __tableau_vers_lignes___api_bitget_v0(char *** tableau, uint T) {
	//0    1    2      3    4    5   6     7      8                 9
	//Unix,Date,Symbol,Open,High,Low,Close,Volume,Volume Base Asset,tradecount
	//float lignes[dar_I][dar_L][dar_N][lignes];
	Ligne_t ** lignes = alloc<Ligne_t*>(5);
	lignes[0] = extraire("prixs"      , tableau, T, 6);
	lignes[1] = extraire("hight"      , tableau, T, 4);
	lignes[2] = extraire("low"        , tableau, T, 5);
	lignes[3] = extraire("volume BTC" , tableau, T, 7);
	lignes[4] = extraire("volume USDT", tableau, T, 8);
	return lignes;
};

//float *** ___eurodollar(char ** tableau, uint lignes);

//	--------------------------------------------------

//	--------------------------------------------------

static uint global_T = 0;

float * ema(float * ligne, float K) {
	float * ret = alloc<float>(global_T);
	ret[0] = ligne[0];
	FOR(1, i, global_T) {
		ret[i] = ret[i-1]*(1.0 - 1.0/K) + ligne[i] * (1.0/K);
	}
	return ret;
};

float * __diff(float * l) {
	float * ret = alloc<float>(global_T);
	ret[0] = 0;
	FOR(1, i, global_T) ret[i] = l[i] - l[i-1];
	return ret;
};

float * __hausse(float * l, float heure) {
	float * ret = alloc<float>(global_T);
	uint _heure = (uint)roundf(heure);
	FOR(0, i, _heure) ret[i] = 0;
	FOR(_heure, i, global_T) ret[i] = l[i] / l[i-_heure] - 1;
	return ret;
};

float * __delta_ema(float * l, float e0, float e1) {
	float * ema_0 = ema(l, e0);
	float * ema_1 = ema(l, e1);
	float * ret = alloc<float>(global_T);
	FOR(0, i, global_T) ret[i] = ema_0[i] - ema_1[i];
	free(ema_1); free(ema_0);
	return ret;
};

void multiplier(float * l, float alpha) {
	FOR(0, i, global_T) l[i] *= alpha;
};

float * __macd(float * l, float K) {
	float * ema12 = ema(l, K*12);
	float * ema26 = ema(l, K*26);
	float * macd = alloc<float>(global_T);
	FOR(0, i, global_T) macd[i] = ema12[i] - ema26[i];
	float * ema9 = ema(macd, K*9);
	float * ret = alloc<float>(global_T);
	FOR(0, i, global_T) ret[i] = macd[i] - ema9[i];
	free(ema12);
	free(ema26);
	free(ema9);
	free(macd);
	return ret;
};

float * __chiffre(float * l, float chiffre) {
	float * ret = alloc<float>(global_T);
	FOR(0, i, global_T) ret[i] = 2*fabs(l[i]/chiffre - roundf(l[i]/chiffre));
	return ret;
};

float * __rsi(float * l, uint n) {
	float * deltas = alloc<float>(global_T);//__diff(l);
	FOR(1, i, global_T) deltas[i] = l[i] - l[i-1];
	//
	/*float * gains  = alloc<float>(global_T);
	float * pertes = alloc<float>(global_T);
	FOR(0, i, global_T) {
		gains [i] = 0;
		pertes[i] = 0;
		if (deltas[i] > 0) gains [i] = deltas[i];
		if (deltas[i] < 0) pertes[i] = -deltas[i];
	}*/
	//
	//float moy_gains  = 0; FOR(0, i, n) moy_gains  += gains [i] / (float)n;
	//float moy_pertes = 0; FOR(0, i, n) moy_pertes += pertes[i] / (float)n;
	//
	float * rsi = alloc<float>(global_T);
	FOR(0, i, n) rsi[i] = 0;
	//
	//rsi[n-1] = (moy_pertes==0 ? 100.0 : (100.0 - (100.0 / (1.0 + (moy_gains/moy_pertes)))));
	//
	FOR(n, i, global_T) {
		//moy_gains  = (moy_gains  - gains [i-n]/(float)n + gains [i]/(float)n);
		//moy_pertes = (moy_pertes - pertes[i-n]/(float)n + pertes[i]/(float)n);
		//
		float mg=0, mp=0;
		FOR(0, j, n) (deltas[i-j]>0?mg:mp) += fabs(deltas[i-j]);
		//
		float rsi_val = 100.0;
		if (mp != 0) {
			rsi_val = 100.0 - 100.0/(1+mg/mp);//(100.0 / (1.0 + moy_gains/moy_pertes));
		}
		rsi[i] = rsi_val;// / 100.0;
	};
	//
	free(deltas);
	//free(gains);
	//free(pertes);
	//
	return rsi;
};

float * __stoch_rsi(float * l, uint n) {
	float * rsi = __rsi(l, n);
	//
	float * ret = alloc<float>(global_T);
	FOR(0, i, global_T) ret[i] = 0;
	//
	FOR(n, i, global_T) {
		float rsi_min=rsi[i], rsi_max=rsi[i];
		FOR(1, j, n) {
			float val = rsi[i-j];
			if (val < rsi_min) rsi_min = val;
			if (val > rsi_max) rsi_max = val;
		};
		//
		if (rsi_max==rsi_min) ret[i] = 0;
		else {
			ret[i] = (rsi[i] - rsi_min) / (rsi_max - rsi_min);
		}
	};
	free(rsi);
	return ret;
};

float * volBU(float * vbtc, float * vusdt, float * prix_btc_en_usdt) {
	float * ret = alloc<float>(global_T);
	FOR(0, i, global_T) {
		float a_moins_b = vbtc[i]*prix_btc_en_usdt[i] - vusdt[i];
		float a_plus__b = (vbtc[i]*prix_btc_en_usdt[i]+vusdt[i])/2;
		ret[i] = a_moins_b/a_plus__b;
	}
	return ret;
};

float * __log(float * f) {
	float * ret = alloc<float>(global_T);
	FOR(0, i, global_T) {
		ret[i] = logf(fabs(f[i]))*(f[i]>=0 ? 1:-1);
	}
	return ret;
};

void __lignes_vers_ema___api_bitget_v0(Ligne_t ** lignes, uint N, char * dar) {
	ASSERT(strcmp(lignes[0]->nom, "prixs"      ) == 0);
	ASSERT(strcmp(lignes[1]->nom, "hight"      ) == 0);
	ASSERT(strcmp(lignes[2]->nom, "low"        ) == 0);
	ASSERT(strcmp(lignes[3]->nom, "volume BTC" ) == 0);
	ASSERT(strcmp(lignes[4]->nom, "volume USDT") == 0);
	//
	uint T = lignes[0]->T;
	global_T = T;
	//
	uint I = 4;	//	Intervs
	uint INTERVS[4] = {1,4,16,64};
	//
	uint L = 13;
	//
#define MAX_INTERV_MULTIPLE 1
	//
	float * lignes_interv[I][L];
	//
	FOR(0, i, I) {
		float * prixs    = ema(lignes[0]->ligne, (float)INTERVS[i]);
		float * hight    = ema(lignes[1]->ligne, (float)INTERVS[i]);
		float * low      = ema(lignes[2]->ligne, (float)INTERVS[i]);
		float * vol_BTC  = ema(lignes[3]->ligne, (float)INTERVS[i]);
		float * vol_USDT = ema(lignes[4]->ligne, (float)INTERVS[i]);
		//
		//
		float heure = (float)INTERVS[i];
		//
		float * prixs1      = __hausse(            prixs         ,heure);  multiplier(prixs1,  25.0);
		float * prixs4      = __hausse(        ema(prixs, 4.0   ),heure);  multiplier(prixs4, 25.0);
		float * delta_26_12 =         (__delta_ema(prixs, 26, 12));  multiplier(delta_26_12, 0.0007);
		float * delta_13_6  =         (__delta_ema(prixs, 13,  6));  multiplier(delta_13_6, 0.0005);
		//
		float * macd1       =         (__macd     (prixs, 1     ));  multiplier(macd1, 0.001);
		float * macd4       =    __log(__macd     (prixs, 4     ));  multiplier(macd4, 0.25);
		//
		float * chiffre1k   =         (__chiffre  (prixs, 1000 ));   multiplier(chiffre1k, 1.0);
		float * chiffre10k  =         (__chiffre  (prixs, 10000));   multiplier(chiffre10k, 1.0);
		//
		float * rsi14       =         (__rsi      (prixs, 14     )); multiplier(rsi14, 0.01);
		float * stoch_rsi14 =         (__stoch_rsi(prixs, 14     )); multiplier(stoch_rsi14, 1.0);
		//
		float * volume_A    = __hausse(        ema(vol_BTC ,10),heure);    multiplier(volume_A, 2.0);
		float * volume_B    = __hausse(        ema(vol_USDT,10),heure);    multiplier(volume_B, 2.0);
		float * volume_AB   = ema(volBU(vol_BTC,vol_USDT,prixs),10); multiplier(volume_AB, 500);
		//
		//
		lignes_interv[i][ 0] = prixs1;
		lignes_interv[i][ 1] = prixs4;
		lignes_interv[i][ 2] = delta_26_12;
		lignes_interv[i][ 3] = delta_13_6;
		lignes_interv[i][ 4] = macd1;
		lignes_interv[i][ 5] = macd4;
		lignes_interv[i][ 6] = chiffre1k;
		lignes_interv[i][ 7] = chiffre10k;
		lignes_interv[i][ 8] = rsi14;
		lignes_interv[i][ 9] = stoch_rsi14;
		lignes_interv[i][10] = volume_A;
		lignes_interv[i][11] = volume_B;
		lignes_interv[i][12] = volume_AB;
	}
	MSG("Lignes écrites !");
	//
	uint DEPART = MAX_INTERV_MULTIPLE * INTERVS[I-1] * N;
	//
	//##########################################################
	//###############  Ecrire le dar.bin  ######################
	//
	FILE * fp = fopen(dar, "wb");
	//
	uint X = I * L * N;
	uint Y = 1 * L * 1;
	//
	uint T_DEPART = T - DEPART;
	FWRITE(&T_DEPART, sizeof(uint), 1, fp);
	FWRITE(&X, sizeof(uint), 1, fp);
	FWRITE(&Y, sizeof(uint), 1, fp);
	FWRITE(&L, sizeof(uint), 1, fp);
	FWRITE(&N, sizeof(uint), 1, fp);
	//
	FWRITE(lignes[0]->ligne+DEPART, sizeof(float), T-DEPART, fp);	//prixs
	//
	//	x__d
	FOR(DEPART, t, T)
		FOR(0, i, I)
			RETRO_FOR(0, n, N)
				FOR(0, l, L)
					FWRITE(&lignes_interv[i][l][t - n*INTERVS[i]], sizeof(float), 1, fp);
	//
	//	y__d
	float zero = 0;
	FOR(DEPART, t, T) {
		FOR(0, l, L) {
			if (t == T-1) {FWRITE(&zero,                                sizeof(float), 1, fp);}
			else          {FWRITE(&lignes_interv[0][l][t+1*INTERVS[0]], sizeof(float), 1, fp);}
		}
	}
	//
	fclose(fp);
	MSG("Dar.bin écrit !");
};

//	--------------------------------------------------

void btcusdt_csv(char * csv, char * dar) {
	//	CSV.csv -> tableau -> Ligne_t* -> Transformations_t* -> dar.bin
	//	-- Lire Fichier ---
	FILE * fp = fopen(csv, "r");
	char * txt = lire_chars(fp);
	fclose(fp);

	uint T = nombre_lignes(txt) - 2;

	char *** tableau = tableau_csv(txt);
	MSG("CSV étrait !");

	//	-- Lire les lignes --
	uint N = 32;
	__lignes_vers_ema___api_bitget_v0(
		__tableau_vers_lignes___api_bitget_v0(tableau, T),
		N,
		dar
	);
}