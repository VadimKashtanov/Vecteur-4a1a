#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__pourcent_btcusdt_stricte(
	float * somme,
	float * potentiel,
	//
	float * prixs,
	float * y, float * w,
	uint * ts__d,
	//
	float coef,
	//
	uint T, uint Y)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	//
	if (_t < GRAND_T) {
		FOR(0, mega_t, MEGA_T) {
			uint ty        = t_MODE(_t, mega_t);
			uint t_btcusdt = ts__d[_t] + mega_t;
			//
			assert(t_btcusdt != T-1);	//	Le dernier est chargé, mais est interdit, car le y__d est nulle (car on connait pas l'heure+1)
			//
			uint wpos = t_btcusdt*Y + 0; float _w = w[wpos];
			uint ypos = ty       *Y + 0; float _y = y[ypos];
			//
			float K = powf(100*fabs(prixs[t_btcusdt+1]/prixs[t_btcusdt] - 1), coef);
			//
			float a_t_il_predit = (float)(sng(_y) == sng(_w));
			//
			atomicAdd(&    somme[0], a_t_il_predit * K);
			atomicAdd(&potentiel[0],       1       * K);
		}
	}
};

float pourcent_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d, float coef) {
	float *     somme__d = cudalloc<float>(1);
	float * potentiel__d = cudalloc<float>(1);
	//
	k__pourcent_btcusdt_stricte<<<dim3(KERD(GRAND_T, 16)), dim3(16)>>>(
		somme__d, potentiel__d,
		//
		btcusdt->prixs__d,
		y__d, btcusdt->y__d,
		ts__d,
		//
		coef,
		//
		btcusdt->T, btcusdt->Y
	);
	ATTENDRE_CUDA();
	//
	float * somme     = gpu_vers_cpu<float>(somme__d, 1);
	float * potentiel = gpu_vers_cpu<float>(potentiel__d, 1);
	float ret = somme[0] / potentiel[0];
	free(somme); free(potentiel);
	//
	return ret;
};
