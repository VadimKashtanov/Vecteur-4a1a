#include "hip/hip_runtime.h"
#include "sous_scal.cuh"

#define BLOQUE 128 //0.30 ms

static __global__ void kerd__sous_scal__simple(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y, 
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint C0)
{
	uint _tc0 = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	uint _c0 = (_tc0 - (_tc0%GRAND_T))/GRAND_T;
	uint _t  = _tc0 - _c0*GRAND_T;

	//if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint tx1 = t_MODE(_t, mega_t-x1_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	uint Vect = X0/C0;
	//
	__shared__ float _sous_;
	if (threadIdx.x == 0 && _y<Vect) _sous_ = x1[tx1*X1 + _c0];
	__syncthreads();

	if (_y < Vect) y[ty*Y + _c0*Vect + _y] = x0[tx0*X0 + _c0*Vect + _y] - _sous_;
};

//	---------------------------------------------------------------------------------

void sous_scal__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint * params = inst->params;
	uint \
		C0 =params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	uint Vect = inst->Y / C0;
	//
	ASSERT(BLOQUE <= Vect);
	//
	if (x0_existe && x1_existe) {
		//FOR(0, _c0, C0) {
			kerd__sous_scal__simple<<<dim3(KERD((GRAND_T*C0),1), KERD(Vect,BLOQUE)), dim3(1,BLOQUE,1)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				inst->x_t[1], inst->x_Y[1], x__d[1],
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				//
		//		_c0,
				//
				C0
			);
		//}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};