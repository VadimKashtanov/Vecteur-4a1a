#include "hip/hip_runtime.h"
#include "sous_scal.cuh"

#define BLOQUE 128

static __global__ void d_kerd__sous_scal__simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint C0)
{
	uint _tc0 = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y   = threadIdx.y + blockIdx.y * blockDim.y;

	uint _c0 = (_tc0 - (_tc0%GRAND_T))/GRAND_T;
	uint _t  = _tc0 - _c0*GRAND_T;

	//if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint tx1 = t_MODE(_t, mega_t-x1_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	uint Vect = X0/C0;
	//
	__shared__ float _d_sous_;
	if (threadIdx.x == 0 && _y<Vect) _d_sous_ = 0;
	__syncthreads();

	if (_y < Vect) {
		float _dy = dy[ty*Y + _c0*Vect + _y];
		//y[ty*Y + _c0*Vect + _y] = x0[tx0*X0 + _c0*Vect + _y] - _sous_;
		atomicAdd(&dx0[tx0*X0 + _c0*Vect + _y], _dy);
		atomicAdd(&_d_sous_, -_dy);
	}
	__syncthreads();

	if (threadIdx.x == 0 && _y<Vect) atomicAdd(&dx1[tx1*X1 + _c0], _d_sous_);
};

//	---------------------------------------------------------------------------------

void sous_scal__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		C0 =params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	uint Vect = inst->Y / C0;
	//
	ASSERT(BLOQUE <= Vect);
	//
	if (x0_existe && x1_existe) {
		//FOR(0, _c0, C0) {
			d_kerd__sous_scal__simple<<<dim3(KERD((GRAND_T*C0),1), KERD(Vect,BLOQUE)), dim3(1,BLOQUE,1)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
		//		_c0,
				//
				C0
			);
		//}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};