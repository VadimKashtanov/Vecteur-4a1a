#include "hip/hip_runtime.h"
#include "matmul_poid_AP.cuh"

#define BLK 8
#define BLK_T 4

static __global__ void kerd__matmul_poid_AP__simple(
	uint x0_t, uint X0, float * x0,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x;
	uint thy = threadIdx.y;
	uint thz = threadIdx.z;
	//
	uint _bx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _ay = threadIdx.y + blockIdx.y * blockDim.y;
	uint _tc0  = threadIdx.z + blockIdx.z * blockDim.z;

	uint _t  = _tc0 % GRAND_T;
	uint _c0 = (_tc0-_t)/GRAND_T;

	__shared__ float A[BLK_T][BLK][BLK];
	__shared__ float P       [BLK][BLK];

	//if (_bx < Bx && _ay < Ay && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		FOR(0, z, KERD(Ax,BLK)) {
			//
			uint _A_y = _ay;//(z*BLK_AX + thy);
			uint _A_x = (z*BLK + thx);
			A[thz][thy][thx] = ((_A_y<Ay && _A_x<Ax) ? x0[tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _A_y*Ax + _A_x] : 0.0);//_ay*Ax + k];
			if (thz==0) {
				uint _P_y = (z*BLK + thy);
				uint _P_x = _bx;//(z*BLK_AX + thx);
				P[thy][thx] = ((_P_y<Ax && _P_x<Bx) ? p[_c0*(Bx*Ax) + _P_y*Bx + _P_x] : 0.0);//k*Bx + _bx];
			}
			//
			__syncthreads();
			//
			FOR(0, k, BLK) s += A[thz][thy][k] * P[k][thx];
			__syncthreads();
		}
		//
	if (_bx < Bx && _ay < Ay && _t < GRAND_T)
		y[ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx] = s;
	//}
};

//	---------------------------------------------------------------------------------

void matmul_poid_AP__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	//ASSERT(x0_existe);
	//
	if (x0_existe) {
		//FOR(0, c0, C0) {
			kerd__matmul_poid_AP__simple<<<dim3(KERD(Bx, BLK), KERD(Ay, BLK), KERD(GRAND_T*C0, BLK_T)), dim3(BLK,BLK,BLK_T)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->p__d,
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				//
		//		c0,
				//
				Ax, Ay, Bx, C0
			);
		//}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};