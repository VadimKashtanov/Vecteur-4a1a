#include "hip/hip_runtime.h"
#include "matmul_poid_AP.cuh"

#define BLK 8
#define BLK_T 4

static __global__ void d_kerd__matmul_poid_AP__simple__dX(
	//dx = dY @ p.T
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x;
	uint thy = threadIdx.y;
	uint thz = threadIdx.z;
	//
	uint _ax = threadIdx.x + blockIdx.x * blockDim.x;
	uint _ay = threadIdx.y + blockIdx.y * blockDim.y;
	uint _tc0  = threadIdx.z + blockIdx.z * blockDim.z;
	//
	uint _t  = _tc0 % GRAND_T;
	uint _c0 = (_tc0-_t)/GRAND_T;
	//
	__shared__ float DY[BLK_T][BLK][BLK];
	__shared__ float Pt       [BLK][BLK];
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	float s = 0;
	FOR(0, z, KERD(Bx, BLK)) {
		uint _DY_x = z*BLK + thx;//_ax;
		uint _DY_y = _ay;//z*BLK + thx;
		DY[thz][thy][thx] = ((_DY_x<Bx && _DY_y<Ay) ? dy[ty*C0*Bx*Ay + _c0*Bx*Ay + _DY_y*Bx+_DY_x]:0);
		uint _Pt_x = _ax;//z*BLK + thy;
		uint _Pt_y = z*BLK + thy;//_bx;
		if (thz == 0)
			Pt[thy][thx] = ((_Pt_x<Ax && _Pt_y<Bx) ? p[_c0*Bx*Ax + TRANSPOSE(_Pt_x,_Pt_y,Ax,Bx)]:0);
		__syncthreads();

		FOR(0, k, BLK) s += DY[thz][thy][k] * Pt[k][thx];
		__syncthreads();
	};

	if (_ax < Ax && _ay < Ay) {
		atomicAdd(&dx0[tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _ay*Ax + _ax], s);
	}
};

static __global__ void d_kerd__matmul_poid_AP__simple__dP(
	//dp = x.T @ dY
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x;
	uint thy = threadIdx.y;
	uint thz = threadIdx.z;
	//
	uint _bx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _ax = threadIdx.y + blockIdx.y * blockDim.y;
	uint _tc0  = threadIdx.z + blockIdx.z * blockDim.z;
	//
	uint _t  = _tc0 % GRAND_T;
	uint _c0 = (_tc0-_t)/GRAND_T;
	//
	__shared__ float Xt[BLK_T][BLK][BLK];
	__shared__ float DY[BLK_T][BLK][BLK];
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	float s = 0;
	FOR(0, z, KERD(Ay,BLK)) {
		uint _Xt_x = _ax;
		uint _Xt_y = z*BLK + thx;
		Xt[thz][thy][thx] = ((_Xt_x<Ax && _Xt_y<Ay) ? x0[tx0*C0*Ax*Ay + _c0*Ax*Ay + _Xt_y*Ax+_Xt_x/*TRANSPOSE(_Xt_x,_Xt_y, Ax,Ay)*/]:0);
		uint _DY_x = _bx;//z*BLK + thy;
		uint _DY_y = z*BLK + thy;//_bx;
		DY[thz][thy][thx] = ((_DY_x<Bx && _DY_y<Ay) ? dy[ty *C0*Bx*Ay + _c0*Bx*Ay + _DY_y*Bx + _DY_x]:0);
		__syncthreads();

		FOR(0, k, BLK) s += Xt[thz][thy][k] * DY[thz][k][thx];
		__syncthreads();
	};

	if (_ax < Ax && _bx < Bx) {
		atomicAdd(&dp[_c0*(Bx*Ax) + _ax*Bx + _bx], s);
	}
};

//	---------------------------------------------------------------------------------

void matmul_poid_AP__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	//ASSERT(x0_existe);
	//
	if (x0_existe) {
		//FOR(0, _c0, C0) {
			d_kerd__matmul_poid_AP__simple__dX<<<dim3(KERD(Ax, BLK), KERD(Ay, BLK), KERD(GRAND_T*C0, BLK_T)), dim3(BLK,BLK,BLK_T)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->p__d, inst->dp__d,
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
			//	_c0,
				//
				Ax, Ay, Bx, C0
			);
			d_kerd__matmul_poid_AP__simple__dP<<<dim3(KERD(Bx, BLK), KERD(Ax, BLK), KERD(GRAND_T*C0, BLK_T)), dim3(BLK,BLK,BLK_T)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->p__d, inst->dp__d,
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
			//	_c0,
				//
				Ax, Ay, Bx, C0
			);
		//}
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};