#include "hip/hip_runtime.h"
#include "batch_variance.cuh"

static __global__ void kerd_batch_variance(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		//uint tx1 = t_MODE(_t, mega_t-x1_t);
		//uint ty  = t_MODE(_t, mega_t     );
		//
		float s = x0[tx0*X0 + _x];
		//
		uint c0 = (_x - (_x%(X0/C0))  )/(X0/C0);
		//
		float miu = x1[0*X1 + c0];
		//
		uint LEN_L = X0 / C0;
		atomicAdd(&y[0*Y + c0], powf(s - miu, 2) / (float)(LEN_L*GRAND_T));
	};
}

void batch_variance__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	if (x0_existe && x1_existe) {
		kerd_batch_variance<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};