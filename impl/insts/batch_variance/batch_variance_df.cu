#include "hip/hip_runtime.h"
#include "batch_variance.cuh"

static __global__ void d_kerd_batch_variance(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y,
	float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		//uint tx1 = t_MODE(_t, mega_t-x1_t);
		//uint ty  = t_MODE(_t, mega_t     );
		//
		uint c0 = (  _x - (_x%(X0/C0))  )/(X0/C0);
		//
		float s = x0[tx0*X0 + _x];
		//
		float miu = x1[0*X1 + c0];
		//
		uint LEN_L = X0 / C0;
		//atomicAdd(&y[0*Y + c0], powf(s - miu, 2) / (float)LEN_L);
		float _dx = 2*(s - miu) / (float)(LEN_L*GRAND_T);
		float _dm = -_dx;
		atomicAdd(&dx0[tx0*X0 + _x], dy[0*Y + c0] * _dx);
		atomicAdd(&dx1[0  *X1 + c0], dy[0*Y + c0] * _dm);
	};
}

void batch_variance__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	if (x0_existe && x1_existe) {
		d_kerd_batch_variance<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
			//
			inst->Y,
			inst->y__d,
			inst->dy__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		// rien
	}
};