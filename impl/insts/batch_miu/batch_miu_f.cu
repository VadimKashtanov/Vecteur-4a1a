#include "hip/hip_runtime.h"
#include "batch_miu.cuh"

static __global__ void kerd_batch_miu(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		//uint ty  = t_MODE(_t, mega_t     );
		//
		float s = x0[tx0*X0 + _x];
		//
		uint LEN_L = X0 / C0;
		//
		uint c0 = ( _x - (_x % LEN_L) )/LEN_L;
		assert(c0 < C0);
		//
		//if (s != s) {
			//printf("tx0=%i _x=%i X0=%i c0=%i C0=%i %f %f %f\n", tx0, _x, X0, c0, C0, x0[tx0*X0 + _x-1], s, x0[tx0*X0 + _x+1]);
			//assert(0);
		//}
		//float somme = 0;
		/*FOR(0, _t, GRAND_T) {
			uint tx0 = t_MODE(_t, mega_t-x0_t);
			somme += x0[tx0*X0 + _x];
			if (x0[tx0*X0 + _x] != x0[tx0*X0 + _x]) printf("%i %i %f\n", tx0, _x, x0[tx0*X0 + _x]);
			//assert(x0[tx0*X0 + _x] != x0[tx0*X0 + _x]);
		}*/
		//y[0*Y + c0] = somme / (float)(LEN_L * GRAND_T);
		atomicAdd(&y[0*Y + c0], s / (float)(LEN_L * GRAND_T));
	};
}

void batch_miu__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	ASSERT(x0_existe);
	//
	if (x0_existe) {
		kerd_batch_miu<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};