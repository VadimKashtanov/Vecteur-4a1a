#include "hip/hip_runtime.h"
#include "drop_vecteur.cuh"

static __global__ void kerd_drop_vecteur(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint VECT, uint POURCENT,
	//
	uint graine,
	uint entrainnement)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint graine_vectorielle = graine + (_y - (_y%VECT))/VECT;
		//
		uint valeur_pseudo_rnd = pseudo_rnd(graine_vectorielle) % 100;
		//
		y[ty*Y + _y] = ((valeur_pseudo_rnd<POURCENT && entrainnement) ? 0.0 : x0[tx0*X0 + _y]);
	};
}

void drop_vecteur__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint \
		VECT     = inst->params[0],	\
		POURCENT = inst->params[1];
	//
	uint graine = ((uint*)inst->espace_potentiel)[0];
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd_drop_vecteur<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			VECT, POURCENT,
			//
			graine,
			entrainnement
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};