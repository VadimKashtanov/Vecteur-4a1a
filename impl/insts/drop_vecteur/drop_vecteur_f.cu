#include "hip/hip_runtime.h"
#include "drop_vecteur.cuh"

static __global__ void kerd_drop_vecteur(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint VECT, uint POURCENT,
	//
	float * matrice,
	uint entrainnement)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		if (entrainnement) {
			/*if (matrice[_y] == 0.0) {
				y[ty*Y + _y] = 0.0;
			} else {
				y[ty*Y + _y] = x0[tx0*X0 + _y];
			}*/
			y[ty*Y + _y] = x0[tx0*X0 + _y] * matrice[_y];
		} else {
			y[ty*Y + _y] = x0[tx0*X0 + _y];
		}
	};
}

void drop_vecteur__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint \
		VECT     = inst->params[0],	\
		POURCENT = inst->params[1];
	//
	float * matrice = (float*)inst->espace_potentiel;
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd_drop_vecteur<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			VECT, POURCENT,
			//
			matrice,
			entrainnement
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};