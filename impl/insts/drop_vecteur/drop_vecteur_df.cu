#include "hip/hip_runtime.h"
#include "drop_vecteur.cuh"

static __global__ void d_kerd_drop_vecteur(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint VECT, uint POURCENT,
	//
	uint graine)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint graine_vectorielle = graine + (_y - (_y%VECT))/VECT;
		//
		uint valeur_pseudo_rnd = pseudo_rnd(graine_vectorielle) % 100;
		//
		if (valeur_pseudo_rnd < POURCENT) {
			//
		} else {
			//y[ty*Y + _y] = x0[tx0*X0 + _y];
			atomicAdd(&dx0[tx0*X0 + _y], dy[ty*Y + _y]);
		}
	};
}

void drop_vecteur__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint \
		VECT     = inst->params[0],	\
		POURCENT = inst->params[1];
	//
	uint graine = ((uint*)inst->espace_potentiel)[0];
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd_drop_vecteur<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			VECT, POURCENT,
			//
			graine
		);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};