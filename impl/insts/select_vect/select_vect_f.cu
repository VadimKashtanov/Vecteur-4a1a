#include "hip/hip_runtime.h"
#include "select_vect.cuh"

__global__
static void kerd__select_vect(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint Vect, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = x0[tx0*X0 + N*Vect + _y];
		//
		y[ty*Y + _y] = s;
	};
};

void select_vect__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint \
		Vect = inst->params[0],	\
		N    = inst->params[1];
	//
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__select_vect<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			Vect, N
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};