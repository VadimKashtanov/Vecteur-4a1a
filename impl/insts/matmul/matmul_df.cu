#include "hip/hip_runtime.h"
#include "matmul.cuh"

/*static __global__ void d_kerd__matmul__simple(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ay*C0
	uint _ay = thx % Ay;
	uint _c0 = (thx-_ay)/Ay;

	//	thy = Bx*GRAND_T
	uint _bx = thy % Bx;
	uint  _t = (thy-_bx)/Bx;

	if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint pos_y = ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx;
		float _dy = dy[pos_y];
		//
		FOR(0, k, Ax) {
			uint pos_x0 = tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _ay*Ax + k;
			uint pos_x1 = tx1*C0*Bx*Ax + _c0*(Bx*Ax) + k*Bx + _bx;
			//
			//s += x0[pos_x0] * x1[pos_x0];
			atomicAdd(&dx0[pos_x0], x1[pos_x1] * _dy);
			atomicAdd(&dx1[pos_x1], x0[pos_x0] * _dy);
		}
	}
};*/

#define BLK 8
#define BLK_T 4

static __global__ void d_kerd__matmul__simple__dA(
	//dx = dY @ p.T
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x;
	uint thy = threadIdx.y;
	uint thz = threadIdx.z;
	//
	uint _ax = threadIdx.x + blockIdx.x * blockDim.x;
	uint _ay = threadIdx.y + blockIdx.y * blockDim.y;
	uint _tc0  = threadIdx.z + blockIdx.z * blockDim.z;
	//
	uint _t  = _tc0 % GRAND_T;
	uint _c0 = (_tc0-_t)/GRAND_T;
	//
	__shared__ float DY[BLK_T][BLK][BLK];
	__shared__ float Bt[BLK_T][BLK][BLK];
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint tx1 = t_MODE(_t, mega_t-x1_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	float s = 0;
	FOR(0, z, KERD(Bx, BLK)) {
		uint _DY_x = z*BLK + thx;//_ax;
		uint _DY_y = _ay;//z*BLK + thx;
		DY[thz][thy][thx] = ((_DY_x<Bx && _DY_y<Ay) ? dy[ty*C0*Bx*Ay + _c0*Bx*Ay + _DY_y*Bx+_DY_x]:0);
		uint _Bt_x = _ax;//z*BLK + thy;
		uint _Bt_y = z*BLK + thy;//_bx;
		Bt[thz][thy][thx] = ((_Bt_x<Ax && _Bt_y<Bx) ? x1[tx1*C0*Bx*Ax + _c0*Bx*Ax + TRANSPOSE(_Bt_x,_Bt_y,Ax,Bx)]:0);
		__syncthreads();

		FOR(0, k, BLK) s += DY[thz][thy][k] * Bt[thz][k][thx];
		__syncthreads();
	};

	if (_ax < Ax && _ay < Ay) {
		atomicAdd(&dx0[tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _ay*Ax + _ax], s);
	}
};

static __global__ void d_kerd__matmul__simple__dB(
	//dp = x.T @ dY
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x;
	uint thy = threadIdx.y;
	uint thz = threadIdx.z;
	//
	uint _bx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _ax = threadIdx.y + blockIdx.y * blockDim.y;
	uint _tc0  = threadIdx.z + blockIdx.z * blockDim.z;
	//
	uint _t  = _tc0 % GRAND_T;
	uint _c0 = (_tc0-_t)/GRAND_T;
	//
	__shared__ float At[BLK_T][BLK][BLK];
	__shared__ float DY[BLK_T][BLK][BLK];
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint tx1 = t_MODE(_t, mega_t-x1_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	float s = 0;
	FOR(0, z, KERD(Ay,BLK)) {
		uint _At_x = _ax;
		uint _At_y = z*BLK + thx;
		At[thz][thy][thx] = ((_At_x<Ax && _At_y<Ay) ? x0[tx0*C0*Ax*Ay + _c0*Ax*Ay + _At_y*Ax+_At_x/*TRANSPOSE(_Xt_x,_Xt_y, Ax,Ay)*/]:0);
		uint _DY_x = _bx;//z*BLK + thy;
		uint _DY_y = z*BLK + thy;//_bx;
		DY[thz][thy][thx] = ((_DY_x<Bx && _DY_y<Ay) ? dy[ty *C0*Bx*Ay + _c0*Bx*Ay + _DY_y*Bx + _DY_x]:0);
		__syncthreads();

		FOR(0, k, BLK) s += At[thz][thy][k] * DY[thz][k][thx];
		__syncthreads();
	};

	if (_ax < Ax && _bx < Bx && _t < GRAND_T) {
		atomicAdd(&dx1[tx1*C0*Bx*Ax + _c0*(Bx*Ax) + _ax*Bx + _bx], s);
	}
};

//	---------------------------------------------------------------------------------

void matmul__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	if (x0_existe && x1_existe) {
		/*d_kerd__matmul__simple<<<dim3(KERD((Ay*C0),16), KERD((Bx*GRAND_T),16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			ts__d, mega_t,
			//
			Ax, Ay, Bx, C0
		);*/


			d_kerd__matmul__simple__dA<<<dim3(KERD(Ax, BLK), KERD(Ay, BLK), KERD(GRAND_T*C0, BLK_T)), dim3(BLK,BLK,BLK_T)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
			//	_c0,
				//
				Ax, Ay, Bx, C0
			);
			d_kerd__matmul__simple__dB<<<dim3(KERD(Bx, BLK), KERD(Ax, BLK), KERD(GRAND_T*C0, BLK_T)), dim3(BLK,BLK,BLK_T)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				ts__d, mega_t,
				//
			//	_c0,
				//
				Ax, Ay, Bx, C0
			);
	} else {
		//inst_zero_mega_t(inst, mega_t);
	}
};