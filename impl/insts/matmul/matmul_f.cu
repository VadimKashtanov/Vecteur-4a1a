#include "hip/hip_runtime.h"
#include "matmul.cuh"

/*static __global__ void kerd__matmul__simple(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	uint thy = threadIdx.y + blockIdx.y * blockDim.y;

	//	thx = Ay*C0
	uint _ay = thx % Ay;
	uint _c0 = (thx-_ay)/Ay;

	//	thy = Bx*GRAND_T
	uint _bx = thy % Bx;
	uint  _t = (thy-_bx)/Bx;

	if (_ay < Ay && _c0 < C0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		uint pos_y = ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx;
		FOR(0, k, Ax) {
			uint pos_x0 = tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _ay*Ax + k;
			uint pos_x1 = tx1*C0*Bx*Ax + _c0*(Bx*Ax) + k*Bx + _bx;
			//
			if (x0[pos_x0] != x0[pos_x0] || x1[pos_x1]!=x1[pos_x1]) {
				printf("%f %f %i %i\n", x0[pos_x0], x1[pos_x1], pos_x0, pos_x1);
				assert(0);
			}
			s += x0[pos_x0] * x1[pos_x1];
		}
		y[pos_y] = s;
		assert(s==s);
	}
};*/

#define BLK 8
#define BLK_T 4

static __global__ void kerd__matmul__simple(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	//
	uint    Y,
	float * y,
	//
	uint * ts__d, uint mega_t,
	//
	//uint _c0,
	//
	uint Ax, uint Ay, uint Bx, uint C0)
{
	uint thx = threadIdx.x;
	uint thy = threadIdx.y;
	uint thz = threadIdx.z;
	//
	uint _bx = threadIdx.x + blockIdx.x * blockDim.x;
	uint _ay = threadIdx.y + blockIdx.y * blockDim.y;
	uint _tc0  = threadIdx.z + blockIdx.z * blockDim.z;

	uint _t  = _tc0 % GRAND_T;
	uint _c0 = (_tc0-_t)/GRAND_T;

	__shared__ float A[BLK_T][BLK][BLK];
	__shared__ float B[BLK_T][BLK][BLK];

	//if (_bx < Bx && _ay < Ay && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint tx1 = t_MODE(_t, mega_t-x1_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = 0;
		//
		FOR(0, z, KERD(Ax,BLK)) {
			//
			uint _A_y = _ay;//(z*BLK_AX + thy);
			uint _A_x = (z*BLK + thx);
			A[thz][thy][thx] = ((_A_y<Ay && _A_x<Ax) ? x0[tx0*C0*Ax*Ay + _c0*(Ax*Ay) + _A_y*Ax + _A_x] : 0.0);//_ay*Ax + k];
			uint _B_y = (z*BLK + thy);
			uint _B_x = _bx;//(z*BLK_AX + thx);
			B[thz][thy][thx] = ((_B_y<Ax && _B_x<Bx) ? x1[tx1*C0*Bx*Ax + _c0*(Bx*Ax) + _B_y*Bx + _B_x] : 0.0);//k*Bx + _bx];
			
			//
			__syncthreads();
			//
			FOR(0, k, BLK) s += A[thz][thy][k] * B[thz][k][thx];
			__syncthreads();
		}
		//
	if (_bx < Bx && _ay < Ay && _t < GRAND_T)
		y[ty*Y + _c0*(Bx*Ay) + _ay*(Bx) + _bx] = s;
	//}
};

//	---------------------------------------------------------------------------------

void matmul__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint * params = inst->params;
	uint \
		Ax =params[0],	\
		Ay =params[1],	\
		Bx =params[2],	\
		C0 =params[3];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	//
	ASSERT(x0_existe && x1_existe);
	//
	if (x0_existe && x1_existe) {
		//FOR(0, _c0, C0) {
			//kerd__matmul__simple<<<dim3(KERD((Ay*C0),16), KERD((Bx*GRAND_T),16)), dim3(16,16)>>>(
			kerd__matmul__simple<<<dim3(KERD(Bx, BLK), KERD(Ay, BLK), KERD(GRAND_T*C0, BLK_T)), dim3(BLK,BLK,BLK_T)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				inst->x_t[1], inst->x_Y[1], x__d[1],
				//
				inst->Y,
				inst->y__d,
				//
				ts__d, mega_t,
				//
			//	_c0,
				//
				Ax, Ay, Bx, C0
			);
		//}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};