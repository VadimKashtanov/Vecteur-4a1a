#include "hip/hip_runtime.h"
#include "concatenation.cuh"

__global__
static void kerd__concatenation(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint Ax, uint Ay, uint Ay_c0, uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float _x = x0[tx0*X0 + _y];
		assert(_x == _x);	//concatenation
		//
		uint c0 = (_y - (_y%(Ax*Ay*Ay_c0)))/(Ax*Ay*Ay_c0);
		_y -= c0*Ax*Ay*Ay_c0;
		uint c0_Ay_c0 = (_y - (_y%(Ax*Ay)))/(Ax*Ay);
		_y -= c0_Ay_c0*Ax*Ay;
		uint y_Ay = (_y - (_y%(Ax)))/(Ax);
		_y -= y_Ay*Ax;
		uint x_Ax = (_y - (_y%1))/1;
		_y -= x_Ax;
		//
		assert(_y == 0);
		//
		y[ty*Y + c0*Ax*Ay*Ay_c0 + y_Ay*(Ay_c0*Ax) + c0_Ay_c0*Ax + x_Ax] = _x;
	};
};

void concatenation__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Ax    = inst->params[0],	\
		Ay    = inst->params[1],	\
		Ay_c0 = inst->params[2],	\
		C0    = inst->params[3];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__concatenation<<<dim3(KERD(X0,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			Ax, Ay, Ay_c0, C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};