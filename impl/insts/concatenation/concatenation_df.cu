#include "hip/hip_runtime.h"
#include "concatenation.cuh"

__global__
static void d_kerd__concatenation(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint Ax, uint Ay, uint Ay_c0, uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint __x = tx0*X0 + _y;
		//float _x = x0[__x];
		//
		uint c0 = (_y - (_y%(Ax*Ay*Ay_c0)))/(Ax*Ay*Ay_c0);
		_y -= c0*Ax*Ay*Ay_c0;
		uint c0_Ay_c0 = (_y - (_y%(Ax*Ay)))/(Ax*Ay);
		_y -= c0_Ay_c0*Ax*Ay;
		uint y_Ay = (_y - (_y%(Ax)))/(Ax);
		_y -= y_Ay*Ax;
		uint x_Ax = (_y - (_y%1))/1;
		_y -= x_Ax;
		//
		assert(_y == 0);
		//
		float _dy = dy[ty*Y + c0*Ax*Ay*Ay_c0 + y_Ay*(Ay_c0*Ax) + c0_Ay_c0*Ax + x_Ax];
		atomicAdd(&dx0[__x], _dy * 1);
	};
};

void concatenation__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Ax    = inst->params[0],	\
		Ay    = inst->params[1],	\
		Ay_c0 = inst->params[2],	\
		C0    = inst->params[3];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__concatenation<<<dim3(KERD(X0,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			Ax, Ay, Ay_c0, C0
		);
	} else {
		//	rien
	}
};