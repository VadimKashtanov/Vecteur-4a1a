#include "hip/hip_runtime.h"
#include "concatenation.cuh"

__global__
static void d_kerd__concatenation(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint Ax, uint Ay, uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		uint _c0 = (_y - (_y % (Ax*Ay))) / (Ax*Ay);
		_y -= _c0*Ax*Ay;
		//
		uint _y_ = (_y - (_y%Ax)) / Ax;
		_y -= _y_*Ax;
		//
		uint _x_ = (_y - 0) / 1;
		_y -= _x_*1;
		//
		assert(_y == 0);
		//
		//y[ty*Y + _y_*(Ax*C0) + _c0*Ax + _y_];
		atomicAdd(&dx0[tx0*X0 + _c0*Ax*Ay + _y_*Ax + _x_], dy[ty*Y + _y_*(Ax*C0) + _c0*Ax + _x_]);
	};
};

void concatenation__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Ax    = inst->params[0],	\
		Ay    = inst->params[1],	\
		C0    = inst->params[2];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__concatenation<<<dim3(KERD(X0,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			Ax, Ay, C0
		);
	} else {
		//	rien
	}
};