#include "batch_norm_scale.cuh"

uint batch_norm_scale__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	uint \
		C0 = params[0];
	return 2 * C0;
};

uint batch_norm_scale__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void batch_norm_scale__init_poids(Inst_t * inst) {
	//uint \
	//	C0 = inst->params[0];
	//
	ASSERT(inst->Y == inst->x_Y[0]);
	//
	float p[inst->P];
	FOR(0, i, inst->P) p[i] = poid_1_1();

	CONTROLE_CUDA(hipMemcpy(inst->p__d, p, sizeof(float)*inst->P, hipMemcpyHostToDevice));
};

void batch_norm_scale__pre_f(Inst_t * inst) {
	
};

void batch_norm_scale__pre_batchique(Inst_t * inst) {
	
};