#include "hip/hip_runtime.h"
#include "batch_norm_scale.cuh"

static __global__ void kerd_batch_norm_scale(
	uint x0_t, uint X0, float * x0,
	uint x1_t, uint X1, float * x1,
	uint x2_t, uint X2, float * x2,
	//
	float * p,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		//uint tx1 = t_MODE(_t, mega_t-x1_t);
		//uint tx2 = t_MODE(_t, mega_t-x2_t);
		uint ty  = t_MODE(_t, mega_t);
		//
		float s = x0[tx0*X0 + _x];
		//
		uint c0 = (_x - (_x%(X0/C0))  )/(X0/C0);
		//
		float miu = x1[0*X1 + c0];
		float var = x2[0*X2 + c0];
		//
		float alpha = p[c0*2 + 0];
		float beta  = p[c0*2 + 1];
		//
		y[ty*Y + _x] = alpha * (s - miu) / sqrtf(var + 1e-8) + beta;
	};
}

void batch_norm_scale__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint x2_t = inst->x_t[2];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	bool x2_existe = (mega_t != 0 ? true : (x2_t != 1));
	//
	ASSERT(x0_existe && x1_existe && x2_existe);
	//
	if (x0_existe && x1_existe && x2_existe) {
		kerd_batch_norm_scale<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1],
			inst->x_t[2], inst->x_Y[2], x__d[2],
			//
			inst->p__d,
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};