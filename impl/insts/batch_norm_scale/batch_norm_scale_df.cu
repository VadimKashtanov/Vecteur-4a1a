#include "hip/hip_runtime.h"
#include "batch_norm_scale.cuh"

static __global__ void d_kerd_batch_norm_scale(
	uint x0_t, uint X0, float * x0, float * dx0,
	uint x1_t, uint X1, float * x1, float * dx1,
	uint x2_t, uint X2, float * x2, float * dx2,
	//
	float * p, float * dp,
	//
	uint    Y,
	float * y,
	float * dy,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _x = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_x < X0 && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		//uint tx1 = t_MODE(_t, mega_t-x1_t);
		//uint tx2 = t_MODE(_t, mega_t-x2_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float s = x0[tx0*X0 + _x];
		//
		uint c0 = (_x - (_x%(X0/C0))  )/(X0/C0);
		//
		float miu = x1[0*X1 + c0];
		float var = x2[0*X2 + c0];
		//
		float alpha = p[c0*2 + 0];
		//float beta  = p[c0*2 + 1];
		//
		float _dx = alpha * 1 / sqrtf(var + 1e-8);
		float _dm = -_dx;
		float _dv = -alpha * (s - miu) / (2*powf(var + 1e-8, 1.5));
		//
		float dalpha = (s - miu) / sqrtf(var + 1e-8);
		float dbeta  = 1;
		//
		float _dy = dy[ty*Y + _x];
		//
		atomicAdd(&dx0[tx0*X0 + _x], _dy * _dx);
		atomicAdd(&dx1[0  *X1 + c0], _dy * _dm);
		atomicAdd(&dx2[0  *X2 + c0], _dy * _dv);
		//
		atomicAdd(&dp [c0*2 + 0], _dy * dalpha);
		atomicAdd(&dp [c0*2 + 1], _dy * dbeta );
	};
}

void batch_norm_scale__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint \
		C0 = inst->params[0];
	//
	uint x0_t = inst->x_t[0];
	uint x1_t = inst->x_t[1];
	uint x2_t = inst->x_t[2];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	bool x1_existe = (mega_t != 0 ? true : (x1_t != 1));
	bool x2_existe = (mega_t != 0 ? true : (x2_t != 1));
	//
	ASSERT(x0_existe && x1_existe && x2_existe);
	//
	if (x0_existe && x1_existe && x2_existe) {
		d_kerd_batch_norm_scale<<<dim3(KERD(inst->x_Y[0],16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			inst->x_t[1], inst->x_Y[1], x__d[1], dx__d[1],
			inst->x_t[2], inst->x_Y[2], x__d[2], dx__d[2],
			//
			inst->p__d, inst->dp__d,
			//
			inst->Y,
			inst->y__d,
			inst->dy__d,
			//
			mega_t,
			//
			C0
		);
	} else {
		// rien
	}
};