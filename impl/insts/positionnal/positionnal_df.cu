#include "hip/hip_runtime.h"
#include "positionnal.cuh"

__global__
static void d_kerd__positionnal(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y,
	float * dy,
	//
	uint mega_t,
	//
	uint L, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );

		//float e = x0[tx0*X0 + _y];

		atomicAdd(&dx0[tx0*X0 + _y], dy[ty*Y + _y] * 1);
	};
};

void positionnal__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		d_kerd__positionnal<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d,
			inst->dy__d,
			//
			mega_t,
			//
			inst->params[0], inst->params[1]
		);
	} else {
		// rien
	}
};