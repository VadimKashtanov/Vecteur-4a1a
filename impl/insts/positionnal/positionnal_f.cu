#include "hip/hip_runtime.h"
#include "positionnal.cuh"

__global__
static void kerd__positionnal(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint L, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float e = x0[tx0*X0 + _y];
		//
		uint i = (_y % N);
		uint k = (i%2==0 ? i/1 : (i-1)/2);
		float a;
		if (i % 2 == 0) a = sin((float)i / powf(10000.0, 2*k/(float)L));
		if (i % 2 == 1) a = cos((float)i / powf(10000.0, 2*k/(float)L));
		//
		y[ty*Y + _y] = e+a;
	};
};

void positionnal__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint x0_t = inst->x_t[0];
	uint Y  = inst->Y;
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		kerd__positionnal<<<dim3(KERD(Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			inst->params[0], inst->params[1]
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};