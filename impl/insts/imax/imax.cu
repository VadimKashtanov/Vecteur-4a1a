#include "hip/hip_runtime.h"
#include "imax.cuh"

uint imax__calculer_P(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

uint imax__calculer_L(uint X[MAX_XS], uint x[MAX_XS], uint t[MAX_XS], uint Y, uint params[MAX_PARAMS]) {
	return 0;
};

void imax__init_poids(Inst_t * inst) {
	ASSERT(inst->Y == inst->params[0]);
	ASSERT(inst->x_Y[0] % inst->params[0] == 0);
	//inst->p__d;
};

static __global__ void _FLT_MIN(
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_y < Y && _t < GRAND_T) {
		uint ty  = t_MODE(_t, mega_t);
		//
		y[ty*Y + _y] = -FLT_MAX;
	}
}

void imax__pre_f(Inst_t * inst) {
	uint C0 = inst->params[0];
	FOR(0, mega_t, MEGA_T) {
		_FLT_MIN<<<dim3(KERD(inst->Y,16), KERD(GRAND_T,16)), dim3(16,16)>>>(
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			C0
		);
	}
};

void imax__pre_batchique(Inst_t * inst) {
	
};