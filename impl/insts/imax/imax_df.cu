#include "hip/hip_runtime.h"
#include "imax.cuh"

#define BLOQUE 128

__global__
static void d_kerd__imax(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint _c0,
	//
	uint C0)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	//if (_y < Y && _t < GRAND_T) {
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	//__shared__ float __x_[128];
	__shared__ float _dy_[1];
	__shared__ float __y_[1];
	//if ( _y < (X0/C0) ) _x_[threadIdx.x] = x0[tx0*X0 + _y];
	//else                _x_[threadIdx.x] = -1e35.f;
	//__syncthreads();
	//
	/*uint lg = (uint)log2f((float)Vect);
	FOR(1, l, lg+1) {
		//uint p = pow(2, l);
		uint p = 1 << l;
		uint p1 = 1 << (l-1);
		if (thx % p == 0) _x_[thx] = MAX2(_x_[thx], _x_[thx+p1]);
		__syncthreads();
	}
	__syncthreads();*/
	//
	if (threadIdx.x == 0 && _y<Y) {
		//atomicMax(&y[ty*Y + _c0], _x_[0]);
		__y_[0] =  y[ty*Y + _c0];
		_dy_[0] = dy[ty*Y + _c0];
	}
	__syncthreads();

	if (x0[tx0*X0 + _y] == __y_[0]) {
		atomicAdd(&dx0[tx0*X0 + _y], _dy_[0]);
	}
};

void imax__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		C0 = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	//printf("%i %i\n", X0,C0);
	ASSERT(BLOQUE <= (X0/C0));
	//
	if (x0_existe) {
		FOR(0, _c0, C0) {
			d_kerd__imax<<<dim3(KERD((X0/C0),BLOQUE),  KERD(GRAND_T,1)), dim3(BLOQUE,1)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
				//
				inst->Y,
				inst->y__d, inst->dy__d,
				//
				mega_t,
				//
				_c0,
				//
				C0
			);
		}
	} else {
		//	rien
	}
};