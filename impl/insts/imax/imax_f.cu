#include "hip/hip_runtime.h"
#include "imax.cuh"

#define BLOQUE 128

__global__
static void kerd__imax(	//Un BLOQUE couvre tout le _c0
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint C0)
{
	uint thx = threadIdx.x;
	//
	uint _c0 = blockIdx.x;
	uint _t  = blockIdx.y;
	//
	//if (_y < Y && _t < GRAND_T) {
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	__shared__ float _x_[BLOQUE];
	__shared__ float _max;
	//
	if (thx == 0) _max = -FLT_MAX;
	//
	//
	FOR(0, partie, KERD((X0/C0), BLOQUE)) {
		//
		uint _y = _c0*(X0/C0) + partie*BLOQUE + thx;
		//
		if ( _y < X0 ) _x_[threadIdx.x] = x0[tx0*X0 + _y];
		else           _x_[threadIdx.x] = -FLT_MAX;
		__syncthreads();
		//
		uint lg = (uint)log2f((float)(X0/C0));
		FOR(1, l, lg+1) {
			//uint p = pow(2, l);
			uint p = 1 << l;
			uint p1 = 1 << (l-1);
			if (threadIdx.x % p == 0) _x_[threadIdx.x] = MAX2(_x_[threadIdx.x], _x_[threadIdx.x+p1]);
			__syncthreads();
		}
		__syncthreads();
		//
		if (threadIdx.x == 0) _max = MAX2(_max, _x_[0]);
	}
	__syncthreads();
	//
	if (threadIdx.x == 0) {
		y[ty*Y + _c0] = _max;
	}
};

void imax__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		C0    = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	if (x0_existe) {
		//FOR(0, _c0, C0) {
			kerd__imax<<<dim3(KERD(/*(X0/*/C0/*)*/,/*BLOQUE*/1), KERD(GRAND_T,1)), dim3(BLOQUE,1)>>>(
				inst->x_t[0], inst->x_Y[0], x__d[0],
				//
				inst->Y,
				inst->y__d,
				//
				mega_t,
				//
				//_c0,
				//
				C0
			);
		//}
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};