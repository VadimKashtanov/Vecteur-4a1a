#include "hip/hip_runtime.h"
#include "softmax.cuh"

__global__
static void kerd__softmax(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint Vect)
{
	uint _v = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint Vects = X0 / Vect;
	//
	if (_v < Vects && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		float max = x0[tx0*X0 + _v*Vect + 0];
		FOR(1, i, Vect) {
			float val = x0[tx0*X0 + _v*Vect + i];
			if (max < val) max = val;
		};
		//
		float somme = 0;
		FOR(0, i, Vect) {
			float val = x0[tx0*X0 + _v*Vect + i];
			assert(val == val);
			somme += expf(x0[tx0*X0 + _v*Vect + i] - max);
		}
		assert(somme == somme);
		//
		FOR(0, i, Vect) {
			float val = expf(x0[tx0*X0 + _v*Vect + i] - max) / somme;
			assert(val == val);	//	Softmax
			y[ty*X0 + _v*Vect + i] = val;
		}
	};
};

void softmax__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Vect = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint Vects = X0 / Vect;
	//
	if (x0_existe) {
		kerd__softmax<<<dim3(KERD(Vects,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			Vect
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};