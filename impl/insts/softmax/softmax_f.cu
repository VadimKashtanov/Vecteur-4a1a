#include "hip/hip_runtime.h"
#include "softmax.cuh"

#define _VECT_ 128

__global__
static void kerd__softmax(
	uint x0_t, uint X0, float * x0,
	//
	uint    Y,
	float * y,
	//
	uint mega_t,
	//
	uint Vect)
{
	extern __shared__ float shared_dynamique[];
	assert(shared_dynamique != 0);
	//
	//	softmax(x) = exp(x-max) / sum(exp(x-max))
	//	1 max
	//	2 exp(x-max) && sum
	//	3 /= sum
	//
	uint thx = threadIdx.x;
	//
	uint _v = blockIdx.x;
	uint _t = blockIdx.y;
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	//
	float * _x_ = shared_dynamique + 0;
	//float * max = shared_dynamique + Vect;
	//float * sum = shared_dynamique + Vect + Vect;
	//__shared__ float _x_[_VECT_];
	//__shared__ float max[_VECT_];
	//__shared__ float sum[_VECT_];
	//
	//
	_x_[thx] = x0[tx0*X0 + _v*Vect + thx];
	__syncthreads();
	//
	//
	//
	//	1) Max
	//max[thx] = _x_[thx];
	//__syncthreads();
	//
	//atomicMax(&max[0], _x_[thx]);
	/*uint lg = (uint)log2f((float)Vect);
	FOR(1, l, lg+1) {
		//uint p = pow(2, l);
		uint p = 1 << l;
		uint p1 = 1 << (l-1);
		if (thx % p == 0) max[thx] = MAX2(max[thx], max[thx+p1]);
		__syncthreads();
	}
	__syncthreads();*/
	//
	//
	//
	//	2) exp(x-max) && sum
	float val = expf(_x_[thx]);//expf(_x_[thx] - max[0]);
	//if (_t==0 && _v==0) printf("%f, ", val);
	//_x_[thx] = val;
#define sum _x_
	sum[thx] = val;
	__syncthreads();
	//
	uint lg = (uint)log2f((float)Vect);
	FOR(1, l, lg+1) {
		//uint p = pow(2, l);
		uint p = 1 << l;
		uint p1 = 1 << (l-1);
		if (thx % p == 0) {
			//printf("%f = %f+%f\n", sum[thx] + sum[thx+p], sum[thx], sum[thx+p]);
			sum[thx] = sum[thx] + sum[thx+p1];
		}
		__syncthreads();
	}
	__syncthreads();
	//if (_t==0 && _v==0 && thx==0) printf("\nsomme = %f\n", _x_[0]);
	//
	//
	//
	//	3) /= sum
	assert(sum[0] == sum[0]);
	//FOR(0, j, 128) printf("somme=%f ", sum[j]);
	//printf("\n");
	//
	val = val / _x_[0];
	y[ty*X0 + _v*Vect + thx] = val;
	assert(val == val);
	//printf("%f\n", val);
	assert(val <= 1 && val >= 0);
};

void softmax__f(Inst_t * inst, float ** x__d, uint * ts__d, uint mega_t, uint entrainnement) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Vect = inst->params[0];
	//
	ASSERT(Vect < 1024);
	ASSERT((Vect & (Vect - 1)) == 0);	//Vect est une puissance de 2
	//
	ASSERT(Vect == _VECT_);
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint Vects = X0 / Vect;
	//
	if (x0_existe) {
		kerd__softmax<<<dim3(Vects, GRAND_T), dim3(Vect,1), (Vect*sizeof(float) + Vect*sizeof(float) + Vect*sizeof(float))>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0],
			//
			inst->Y,
			inst->y__d,
			//
			mega_t,
			//
			Vect
		);
	} else {
		inst_zero_mega_t(inst, mega_t);
	}
};