#include "hip/hip_runtime.h"
#include "softmax.cuh"

__global__
static void d_kerd__softmax(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint Vect)
{
	uint _v = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	if (_v < Y && _t < GRAND_T) {
		uint tx0 = t_MODE(_t, mega_t-x0_t);
		uint ty  = t_MODE(_t, mega_t     );
		//
		/*FOR(0, i, Vect) {
			float _dx0 = 0;
			FOR(0, j, Vect) {
				//float val = expf(x0[tx0*X0 + _v*Vect + i]) / somme;
				//y[tx0*X0 + _v*Vect + i] = expf(x0[tx0*X0 + _v*Vect + i]) / somme;
				//if (i == j) atomicAdd(&dx0[tx0*X0 + _v*Vect + i], y[tx0*X0 + _v*Vect + i]);
				//else        atomicAdd(&dx0[tx0*X0 + _v*Vect + i], y[tx0*X0 + _v*Vect + i]);
				float delta = (float)(i == j);
				_dx0 += y[ty*X0 + _v*Vect + j]*(delta - y[ty*X0 + _v*Vect + i]) * dy[ty*X0 + _v*Vect + j];
			}
			atomicAdd(&dx0[tx0*X0 + _v*Vect + i], _dx0);
		}*/
		float max = x0[tx0*X0 + _v*Vect + 0];
		uint max_i = 0;
		FOR(1, i, Vect) {
			float val = x0[tx0*X0 + _v*Vect + i];
			if (max < val) {
				max = val;
				max_i = i;
			}
		};
		//
		float somme = 0;
		FOR(0, i, Vect) somme += expf(x0[tx0*X0 + _v*Vect + i] - max);
		//
		//
		float d_max   = 0;
		float d_somme = 0;
		FOR(0, i, Vect) {
			//y[ty*X0 + _v*Vect + i] = expf(x0[tx0*X0 + _v*Vect + i] - max) / somme;
			d_somme += dy[ty*X0 + _v*Vect + i] * expf(x0[tx0*X0 + _v*Vect + i] - max) / (somme*somme) * (-1);
			atomicAdd(&dx0[tx0*X0 + _v*Vect + i], dy[ty*X0 + _v*Vect + i] * expf(x0[tx0*X0 + _v*Vect + i]-max) / somme);
			d_max += dy[ty*X0 + _v*Vect + i] * expf(x0[tx0*X0 + _v*Vect + i] - max) / somme * (-1);
		}
		FOR(0, i, Vect) {
			//somme += expf(x0[tx0*X0 + _v*Vect + i] - max);
			atomicAdd(&dx0[tx0*X0 + _v*Vect + i], d_somme * expf(x0[tx0*X0 + _v*Vect + i] - max));
			d_max += d_somme * expf(x0[tx0*X0 + _v*Vect + i] - max) * (-1);
		}
		//
		atomicAdd(&dx0[tx0*X0 + _v*Vect + max_i], d_max);
	};
};

void softmax__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Vect    = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint Vects = X0/Vect;
	//
	if (x0_existe) {
		d_kerd__softmax<<<dim3(KERD(Vects,16), KERD(GRAND_T,8)), dim3(16,8)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			Vect
		);
	} else {
		//	rien
	}
};