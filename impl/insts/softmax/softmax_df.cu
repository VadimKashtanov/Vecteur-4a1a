#include "hip/hip_runtime.h"
#include "softmax.cuh"

#define _VECT_ 128

__global__
static void d_kerd__softmax(
	uint x0_t, uint X0, float * x0, float * dx0,
	//
	uint    Y,
	float * y, float * dy,
	//
	uint mega_t,
	//
	uint Vect)
{
	extern __shared__ float shared_dynamique[];
	assert(shared_dynamique != 0);
	//
	//	softmax(x) = exp(x-max) / sum(exp(x-max))
	//	1 max
	//	2 exp(x-max) && sum
	//	3 /= sum
	//
	uint thx = threadIdx.x;
	//
	uint _v = blockIdx.x;
	uint _t = blockIdx.y;
	//
	uint tx0 = t_MODE(_t, mega_t-x0_t);
	uint ty  = t_MODE(_t, mega_t     );
	//
	//
	/*float * _x_ = shared_dynamique + 0;
	float * max = shared_dynamique + Vect;
	float * sum = shared_dynamique + Vect + Vect;
	//
	float * d_x_ = shared_dynamique + Vect + Vect + Vect;
	float * dmax = shared_dynamique + Vect + Vect + Vect + Vect;
	float * dsum = shared_dynamique + Vect + Vect + Vect + Vect + 1;*/
	float * __y_ = shared_dynamique + 0;
	float * _dy_ = shared_dynamique + Vect;
	float * d_x_ = shared_dynamique + Vect + Vect;
	/*//__shared__ float __x_[_VECT_];
	__shared__ float __y_[_VECT_];
	__shared__ float _dy_[_VECT_];
	//__shared__ float max[_VECT_];
	//__shared__ float sum[_VECT_];
	//
	__shared__ float d_x_[_VECT_];
	//__shared__ float dmax[1];
	//__shared__ float dsum[1];*/
	//
	//
	//__x_[thx] = x0[tx0*X0 + _v*Vect + thx];
	__y_[thx] =  y[ty *Y  + _v*Vect + thx];
	_dy_[thx] = dy[ty *Y  + _v*Vect + thx];
	d_x_[thx] = 0;
	//if (thx==0) dmax[thx] = 0;
	//if (thx==0) dsum[thx] = 0;
	__syncthreads();

	FOR(0, k, _VECT_) {
		d_x_[thx] += __y_[k] * ((k==thx) - __y_[thx]) * _dy_[k];
		__syncthreads();
	}

	atomicAdd(&dx0[tx0*X0 + _v*Vect + thx], d_x_[thx]);

	//

	/*{
		//	1) Max
		max[thx] = _x_[thx];
		__syncthreads();
		//
		//atomicMax(&max[0], _x_[thx]);
		uint lg = (uint)log2f((float)Vect);
		FOR(0, l, lg) {
			//uint p = pow(2, l);
			uint p = 1 << l;
			if (thx % p == 0) max[thx] = MAX2(max[thx], max[thx+p]);
			__syncthreads();
		}
		__syncthreads();
	}

	float val;
	{
		//	2) exp(x-max) && sum
		val = expf(_x_[thx] - max[0]);
		sum[thx] = val;
		__syncthreads();
		//
		uint lg = (uint)log2f((float)Vect);
		FOR(0, l, lg) {
			//uint p = pow(2, l);
			uint p = 1 << l;
			if (thx % p == 0) sum[thx] = sum[thx] + sum[thx+p];
			__syncthreads();
		}
		__syncthreads();
	}

	{
		//	3) /= sum
		assert(sum[0] == sum[0]);
		val /= sum[0];
		assert(val == val);
		//
		//float _val = _x_[thx] / sum[0];
		//float val = y[ty*X0 + _v*Vect + thx];
		float _dy = dy[ty*X0 + _v*Vect + thx];
		float _x__thx = val * sum[0];
		printf("%f %f %f\n", _dy, sum[0], _x__thx);
		d_x_[thx] += _dy * 1 / sum[0];
		float ____ = _dy * (-1)*_x__thx / (sum[0]*sum[0]);
		if (____ != ____) {
			printf("%i %i %f\n", thx, _v, ____);
			assert(0);
		}
		atomicAdd(&dsum[0], ____);
		__syncthreads();
	}

	{
		//	4) deriv : 3) exp(x-max) && sum
		val *= sum[0];
		assert(val == val);
		assert(dsum[0] == dsum[0]);
		//sum[thx] = val;
		//float dval = dsum[0];
		d_x_[thx] += dsum[0] * val * 1;
		atomicAdd(&dmax[ 0 ], dsum[0] * val * (-1));
		__syncthreads();
	}
	
	{
		if (val==max[0]) d_x_[thx] += dmax[0];
		__syncthreads();
	}

	if (d_x_[thx] != d_x_[thx]) {
		assert(dmax[0]==dmax[0]);
		assert(dsum[0]==dsum[0]);
		printf("thx=%i _v=%i d_x_[thx]=%f\n", thx, _v, d_x_[thx]);
		assert(0);
	};
	atomicAdd(&dx0[tx0*X0 + _v*Vect + thx], d_x_[thx]);*/
};

void softmax__df(Inst_t * inst, float ** x__d, float ** dx__d, uint * ts__d, uint mega_t) {
	uint X0 = inst->x_Y[0]; uint x0_t = inst->x_t[0];
	//
	uint \
		Vect    = inst->params[0];
	//
	bool x0_existe = (mega_t != 0 ? true : (x0_t != 1));
	//
	uint Vects = X0/Vect;
	//
	if (x0_existe) {
		d_kerd__softmax<<<dim3(Vects, GRAND_T), dim3(Vect,1), (3*Vect)*sizeof(float)>>>(
			inst->x_t[0], inst->x_Y[0], x__d[0], dx__d[0],
			//
			inst->Y,
			inst->y__d, inst->dy__d,
			//
			mega_t,
			//
			Vect
		);
	} else {
		//	rien
	}
};