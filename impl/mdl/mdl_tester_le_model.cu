#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../impl_template/tmpl_etc.cu"

__global__
static void kerd_p1e5(float * p__d, uint p, float _1E5) {
	p__d[p] += _1E5;
};

static void plus_1e5(float * p__d, uint p, float _1E5) {
	kerd_p1e5<<<1,1>>>(p__d, p, _1E5);
	ATTENDRE_CUDA();
};

//	---------------------------------------------------

void tester_le_model(Mdl_t * mdl, BTCUSDT_t * btcusdt) {
	uint ts[GRAND_T];
	FOR(0, t, GRAND_T) ts[t] = rand() % (btcusdt->T - MEGA_T);
	uint * ts__d = cpu_vers_gpu<uint>(ts, GRAND_T);
	//
	mdl_verif(mdl, btcusdt);
	//
	//
	mdl_allez_retour(mdl, btcusdt, ts__d);
	float * grad_cuda[mdl->insts];
	FOR(0, i, mdl->insts) {
		if (mdl->inst[i]->P > 0)
			grad_cuda[i] = gpu_vers_cpu<float>(mdl->inst[i]->dp__d, mdl->inst[i]->P);
	}
	//
	//
	INIT_CHRONO(s)
	DEPART_CHRONO(s)
	//
	float S = mdl_S(mdl, btcusdt, ts__d);
	//
	float _1E5 = 3e-2;//5e-3;
	uint lp = 0;
	FOR(0, i, mdl->insts) {
		printf("#### INSTRUCTION %i (%s Y=%i) ####\n",
			i, 
			inst_Nom[mdl->inst[i]->ID], mdl->inst[i]->Y
		);
		//
		FOR(0, p, mdl->inst[i]->P) {

			//	f(x + 1e-5)
			plus_1e5(mdl->inst[i]->p__d, p, +_1E5);
			float S1e5 = mdl_S(mdl, btcusdt, ts__d);
			plus_1e5(mdl->inst[i]->p__d, p, -_1E5);

			//	df
			float a = (S1e5 - S)/_1E5;

			//	f'
			float b = grad_cuda[i][p];

			//	vitesse
			float vitesse = (float)(++lp) / VALEUR_CHRONO(s);

			//
			printf("%i| ", p);
			PLUME_CMP(a, b);
			if (b != 0) printf(" (x%+f) ", a/b);
			printf(" (%+f m/s)   inst=%i\n", vitesse, i);
		};
	};
	printf("1E5  === dp\n");
	//
	FOR(0, i, mdl->insts) if (mdl->inst[i]->P > 0) free(grad_cuda[i]);
	//
	cudafree<uint>(ts__d);
};