#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../impl_template/tmpl_etc.cu"

uint hists[] = {
	SGD_____HISTOIRE,
	MOMENT__HISTOIRE,
	RMSPROP_HISTOIRE,
	ADAM____HISTOIRE
};

void opti(
	Mdl_t     *     mdl,
	BTCUSDT_t * btcusdt,
	uint      *   ts__d,
	uint              I,
	uint       tous_les,
	uint        methode,
	float         alpha)
{
	//	Ceci initie tout avant le batch (dropout ...)
	mdl_pre_batch(mdl);


	uint nombre_de_poids = 0;
	FOR(0, i, mdl->insts) nombre_de_poids += mdl->inst[i]->P;


	//	--- Hist ---
	float *** hist = alloc<float**>(hists[methode]);
	FOR(0, h, hists[methode]) {
		hist[h] = alloc<float*>(mdl->insts);
		FOR(0, i, mdl->insts) {
			hist[h][i] = cudalloc<float>(mdl->inst[i]->P);
			// = 0
		}
	}

	//	--- Plume ---
	mdl_plume_grad(mdl, btcusdt, ts__d);
	//
	float _max_abs_grad = 1;//mdl_max_abs_grad(mdl);
	if (_max_abs_grad == 0) ERR("Le grad max est = 0");
	//
	alpha /= _max_abs_grad;
	//
	printf("alpha=%f, max_abs_grad=%f => nouveau alpha=%f  (poids=%i)\n", alpha, _max_abs_grad, alpha / _max_abs_grad, nombre_de_poids);
	//
	//	--- Opti  ---
	//
	FOR(0, i, I) {
		if (i != 0) {
			//	dF(x)
			mdl_allez_retour(mdl, btcusdt, ts__d);

			//	x = x - dx
			if (methode == SGD    ) sgd    (mdl, hist, i, alpha, i);
			if (methode == MOMENT ) moment (mdl, hist, i, alpha, i);
			if (methode == RMSPROP) rmsprop(mdl, hist, i, alpha, i);
			if (methode == ADAM   ) adam   (mdl, hist, i, alpha, i);
		}
		//
		if (i % tous_les == 0) {
			float s = mdl_S(mdl, btcusdt, ts__d);
			//
			float p0 = pourcent_btcusdt(btcusdt, mdl->inst[mdl->sortie]->y__d, ts__d, 0);
			float p1 = pourcent_btcusdt(btcusdt, mdl->inst[mdl->sortie]->y__d, ts__d, 1);
			float p4 = pourcent_btcusdt(btcusdt, mdl->inst[mdl->sortie]->y__d, ts__d, 4);
			//
			printf("%3.i/%3.i score=%f ^0=%f%% ^1=%f%% ^4=%f%%\n", i,I,s, p0, p1, p4);
		};
	};
	//
	//
	FOR(0, h, hists[methode]) {
		FOR(0, i, mdl->insts) {
			cudafree<float>(hist[h][i]);
		}
		free(hist[h]);
	}
	free(hist);
}